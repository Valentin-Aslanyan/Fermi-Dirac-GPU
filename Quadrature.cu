#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <lapacke.h>
#include <cblas.h>
#include <lapacke_mangling.h>
#include <lapacke_utils.h>

/*
Here we demonstrate the calculation of single and double integrals on GPUs using CUDA
We simultaneously carry out 20 of the same integral, with different input parameters
The prefix h_ corresponds to host (namely, CPU)
The prefix d_ corresponds to device (namely, GPU)
Specifically, we evaluate the integrals:
\int_3^5  A \theta^2 + \exp(B\theta) d\theta			(1)
\int_3^5 \int_1^2  A \phi+B\theta^2 d\theta d\phi	(2)
A and B are constants, stored in h_params[] and d_params[]
The limits are stored in h_limits[] and d_limits[]
*/


//The following two functions carry out a single integral (1)
__device__ double d_integrand1(double *d_params,double theta)
{
	double integrand=d_params[0]*theta*theta+exp(d_params[1]*theta);
	return integrand;
}

double h_integrand1(double *h_params,double theta)
{
	double integrand=h_params[0]*theta*theta+exp(h_params[1]*theta);
	return integrand;
}

__global__ void d_single_integral(double *d_params, double *d_limits, double *d_result, double *d_w, double *d_x)
{	
	//Calculate the integrand on each thread
	int tid=threadIdx.x;
	extern  __shared__ double temp[];	//Integrand is stored in shared memory
	double *d_result_temp=&temp[0];
	double lim=d_limits[1]-d_limits[0];
	double theta=d_x[tid]*lim+d_limits[0];
	double integrand=d_integrand1(d_params+blockIdx.x*2,theta);
	d_result_temp[tid]=integrand*d_w[tid]*lim;

	//Perform a standard reduction to integrate
    for (int d = blockDim.x>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (threadIdx.x<d)
	{
		d_result_temp[tid]   += d_result_temp[tid+d];
	}
    }
	__syncthreads();
    if (threadIdx.x==0) 
    {	
	d_result[blockIdx.x]=d_result_temp[tid];
    }
}

//Exact, analytic solution
double h_integral1_exact(double *h_params, double *h_limits)
{
	return h_params[0]*(h_limits[1]*h_limits[1]*h_limits[1]-h_limits[0]*h_limits[0]*h_limits[0])/3.0+(exp(h_params[1]*h_limits[1])-exp(h_params[1]*h_limits[0]))/h_params[1];
}


//Two functions to carry out double integral (2)
__device__ double d_integrand2(double *d_params,double theta,double phi)
{
	double integrand=d_params[0]*phi+d_params[1]*theta*theta;
	return integrand;
}

double h_integrand2(double *h_params,double theta,double phi)
{
	double integrand=h_params[0]*phi+h_params[1]*theta*theta;
	return integrand;
}

//Involves now two reductions and the intermediate results stored in main GPU memory
__global__ void d_double_integral(double *d_params, double *d_limits, double *d_result,double *d_result_part, double *d_w, double *d_x)
{	
	//Calculate the integrand on each thread
	int tid=threadIdx.x, d;
	extern  __shared__ double temp[];
	double *d_result_temp=&temp[0];
	double lim1=d_limits[1]-d_limits[0];
	double lim2=d_limits[3]-d_limits[2];
	double theta=d_x[tid]*lim1+d_limits[0];
	double phi=d_x[blockIdx.x]*lim2+d_limits[2];
	double integrand=d_integrand2(d_params+blockIdx.y*2,theta,phi);
	d_result_temp[tid]=integrand*d_w[tid]*d_w[blockIdx.x]*lim1*lim2;

	//Series of reductions over the first integral
    for (d = blockDim.x>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (threadIdx.x<d)
	{
		d_result_temp[tid]   += d_result_temp[tid+d];
	}
    }
	__syncthreads();
	
	//The results of the first integrals are stored in main GPU memory
    if (threadIdx.x==0) 
    {	
	d_result_part[blockIdx.x+gridDim.x*blockIdx.y]=d_result_temp[tid];
    }
	tid=threadIdx.x+blockDim.x*blockIdx.y;


	//Reduction over the second integral to obtain a value for the full double integral
    if(blockIdx.x==0)
    {
	d_result_temp[threadIdx.x]=d_result_part[tid];
 	   for (d = blockDim.x>>1; d > 0; d >>= 1)
 	   {
 	     __syncthreads(); 
 	     if (threadIdx.x<d)
		{
			d_result_temp[threadIdx.x]   += d_result_temp[threadIdx.x+d];
		}
	    }
		__syncthreads();
	    if (threadIdx.x==0) 
	    {	
		d_result[blockIdx.y]=d_result_temp[threadIdx.x];
	    }
    }
}

//Exact, analytic solution
double h_integral2_exact(double *h_params, double *h_limits)
{
	return h_params[0]*(h_limits[3]*h_limits[3]-h_limits[2]*h_limits[2])*(h_limits[1]-h_limits[0])/2.0+h_params[1]*(h_limits[1]*h_limits[1]*h_limits[1]-h_limits[0]*h_limits[0]*h_limits[0])*(h_limits[3]-h_limits[2])/3.0;
}


//Set up the nodes (x) and weights (w) of Gaussian quadrature, using the Golub Welsch algorithm
//coeffs and vectors are allocated arrays which are required only for solving the tridiagonal matrix equation
void gauss_integration_setup(int datapoints, double *weights, double *x,double *coeffs, double *vectors)
{  int idx;

   x[0]=0.0;
   for (idx=1;idx<datapoints;idx++)
	{
	x[idx]=0.0;
	coeffs[idx-1]=0.5/sqrt(1.0-1.0/(4.0*idx*idx));
	}	

   //dstev finds the eigenvalues and vectors of a symmetric matrix
   LAPACKE_dstev(LAPACK_ROW_MAJOR,'v', datapoints, x, coeffs, vectors, datapoints);

   for (idx=0;idx<datapoints;idx++)
	{
	x[idx]=0.5*(x[idx]+1.0);	//This leads to nodes in the range (0,1)
	weights[idx]=vectors[idx]*vectors[idx];
	}
}

int main(int argc,const char** argv)
{
	//number of nodes for integration
	int h_datapoints=32;

	//Initialisation of GPU
	int device_count=0;
	hipGetDeviceCount(&device_count);
	hipSetDevice(0);	//Run on device 0 by default - can be changed if multiple GPUs etc are present

	//Declare, allocate and calculate nodes and weights for integration
	double *h_x,*h_w,*h_c,*h_v;
	h_x=(double*)malloc(h_datapoints*sizeof(double));		
	h_w=(double*)malloc(h_datapoints*sizeof(double));
	h_c=(double*)malloc((h_datapoints-1)*sizeof(double));	
	h_v=(double*)malloc(h_datapoints*h_datapoints*sizeof(double));
	gauss_integration_setup(h_datapoints,h_w,h_x,h_c,h_v);

	//Copy nodes and weights to GPU
	double *d_x, *d_w;
	hipMalloc((void **)&d_x,sizeof(double)*h_datapoints);
	hipMalloc((void **)&d_w,sizeof(double)*h_datapoints);
	hipMemcpy(d_x,h_x,sizeof(double)*h_datapoints,hipMemcpyHostToDevice);
	hipMemcpy(d_w,h_w,sizeof(double)*h_datapoints,hipMemcpyHostToDevice);

	//Allocate integral-specific constants, limits of integration
	int number_of_integrals=20, idx, idx1, idx2;
	double *h_params, *d_params, *h_lims1, *h_lims2, *d_lims1, *d_lims2, *h_result, *h_result2, *d_result_part, *d_result, h_theta, h_phi, h_lim, h_lim1, h_lim2;
	h_params=(double*)malloc(sizeof(double)*number_of_integrals*2);
	h_lims1=(double*)malloc(sizeof(double)*2);
	h_lims2=(double*)malloc(sizeof(double)*4);
	h_lims1[0]=3.0; h_lims1[1]=5.0;
	h_lims2[0]=3.0; h_lims2[1]=5.0; h_lims2[2]=1.0; h_lims2[3]=2.0;
	h_result=(double*)malloc(sizeof(double)*number_of_integrals*2);
	h_result2=(double*)malloc(sizeof(double)*number_of_integrals*2);
	dim3 grid_dim(h_datapoints,number_of_integrals);
	hipMalloc((void **)&d_params,sizeof(double)*number_of_integrals*2);
	hipMalloc((void **)&d_lims1,sizeof(double)*2);
	hipMalloc((void **)&d_lims2,sizeof(double)*4);
	hipMalloc((void **)&d_result,sizeof(double)*number_of_integrals*2);
	hipMalloc((void **)&d_result_part,sizeof(double)*number_of_integrals*h_datapoints);
	hipMemcpy(d_lims1,h_lims1,sizeof(double)*2,hipMemcpyHostToDevice);
	hipMemcpy(d_lims2,h_lims2,sizeof(double)*4,hipMemcpyHostToDevice);

	//Set A and B parameters of integrals to something non-trivial
	for(int idx=0;idx<number_of_integrals;idx++)
		{
		h_params[idx*2]=0.25*(((double)idx)+1.0);
		h_params[idx*2+1]=0.125*((double)(idx*idx)+1.0);
		}

	//Timing variables
	float d_time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Start integration
	hipEventRecord(start);

	//Copy parameters
	hipMemcpy(d_params,h_params,sizeof(double)*number_of_integrals*2,hipMemcpyHostToDevice);
	//Do both integrals (on default stream in this case)
	d_single_integral<<<number_of_integrals,h_datapoints,h_datapoints*sizeof(double)>>>(d_params,d_lims1,d_result,d_w,d_x);
	d_double_integral<<<grid_dim,h_datapoints,h_datapoints*sizeof(double)>>>(d_params,d_lims2,d_result+number_of_integrals,d_result_part,d_w,d_x);
	//Copy result back
	hipMemcpy(h_result,d_result,sizeof(double)*number_of_integrals*2,hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventElapsedTime(&d_time, start, stop);

	//CPU integral evaluation
	h_lim=h_lims1[1]-h_lims1[0];
	h_lim1=h_lims2[1]-h_lims2[0];
	h_lim2=h_lims2[3]-h_lims2[2];
	for(idx=0;idx<number_of_integrals;idx++)
		{
		h_result2[idx]=0.0;
		h_result2[idx+number_of_integrals]=0.0;
		for(idx1=0;idx1<h_datapoints;idx1++)
			{
			h_theta=h_x[idx1]*h_lim+h_lims1[0];
			h_result2[idx]+=h_integrand1(h_params+idx*2,h_theta)*h_w[idx1]*h_lim;

			h_theta=h_x[idx1]*h_lim1+h_lims2[0];
			for(idx2=0;idx2<h_datapoints;idx2++)
				{
				h_phi=h_x[idx2]*h_lim2+h_lims2[2];
				h_result2[idx+number_of_integrals]+=h_integrand2(h_params+idx*2,h_theta,h_phi)*h_w[idx1]*h_w[idx2]*h_lim1*h_lim2;
				}
			}
		}
	
	//Print result
	printf("GPU time elapsed: %f\n",d_time);
	printf("           Single Integral            |           Double Integral\n");
	printf("    GPU     |    CPU     |   Exact    |    GPU     |    CPU     |   Exact \n");
	for(idx=0;idx<number_of_integrals;idx++)
		{
		printf("%E %E %E %E %E %E\n",h_result[idx],h_result2[idx],h_integral1_exact(h_params+idx*2,h_lims1),h_result[idx+number_of_integrals],h_result2[idx+number_of_integrals],h_integral2_exact(h_params+idx*2,h_lims2));
		}

   //Clean up
   free(h_x);
   free(h_w);
   free(h_c);
   free(h_v);
   free(h_params);
   free(h_lims1);
   free(h_lims2);
   free(h_result);
   hipFree(d_x);
   hipFree(d_w);
   hipFree(d_params);
   hipFree(d_lims1);
   hipFree(d_lims2);
   hipFree(d_result);
   hipFree(d_result_part);
   hipDeviceReset();
}
