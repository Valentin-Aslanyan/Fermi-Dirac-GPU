#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <cblas.h>
extern "C" {  
#include "Rate_Functions_CPU.h"
}
#include "Rate_Functions_GPU.h"

#define ACC_J 19.013
#define ACC_K 25.253
#define ACC_L 6503.0
#define ACC_Jf 19.013f
#define ACC_Kf 25.253f
#define ACC_Lf 6503.0f

//The prefix d_ corresponds to device (namely, GPU)
//This file contains GPU equivalents to the fully-CPU rate calculations in Rate_Functions_CPU.c
//Readers are advised to familiarise themselves with functions in the former file

//Constants that do not change between calculations
__constant__ int d_excitations_number,d_ionizations_number, d_datapoints, d_block_mult;
__constant__ double d_T_r;
__constant__ float d_T_r_f;

//Functions to evaluate cross sections and collision strengths on the GPU
__device__ double d_j_int(double E0,double E_j,double *B_vector)
{
	double Eq=E_j/E0;
	double integrand=-log(Eq)*B_vector[0]+B_vector[1]+Eq*(B_vector[2]+Eq*B_vector[3]);
	return integrand;
}

__device__ double d_k_int(double E0,double E1,double E_i,double *C_vector)
{	double E_i_pow=E_i*E_i, E0_pow=E0*E0, E1_pow=E1*E1, E1_prime=E0-E_i-E1, E1_prime_pow=E1_prime*E1_prime;
	double a=0.5*(sqrt(E0_pow+4.0*E_i_pow)-E0);
	double b=a+E_i;
	double integrand=(1.0/((E1+a)*(E1+b))+1.0/((E1_prime+a)*(E1_prime+b)))*C_vector[0];
	integrand+=2.0*C_vector[1]/E0;
	integrand+=2.0*C_vector[2]*(E0-E_i)/E0_pow; E0_pow*=E0;
	integrand+=3.0*C_vector[3]*(E1_pow+E1_prime_pow)/E0_pow; E0_pow*=E0; E1_pow*=E1; E1_prime_pow*=E1_prime;
	integrand+=4.0*C_vector[4]*(E1_pow+E1_prime_pow)/E0_pow;
	integrand*=0.5/E_i;
	return integrand;
}

__device__ double d_l_int(double EGamma,double T_r,double *D_vector)
{	double exp_EG=exp(EGamma/T_r)-1.0;
	double integrand=(D_vector[0]+D_vector[1]/EGamma)/exp_EG;
	return integrand;
}

__device__ float d_j_int_f(float E0,float E_j,float *B_vector)
{
	float Eq=E_j/E0;
	float integrand=-logf(Eq)*B_vector[0]+B_vector[1]+Eq*(B_vector[2]+Eq*B_vector[3]);
	return integrand;
}


__device__ float d_k_int_f(float E0,float E1,float E_i,float *C_vector)
{	float E_i_pow=E_i*E_i, E0_pow=E0*E0, E1_pow=E1*E1, E1_prime=E0-E_i-E1, E1_prime_pow=E1_prime*E1_prime;
	float a=0.5f*(sqrtf(E0_pow+4.0f*E_i_pow)-E0);
	float b=a+E_i;
	float integrand=(1.0f/((E1+a)*(E1+b))+1.0f/((E1_prime+a)*(E1_prime+b)))*C_vector[0];
	integrand+=2.0f*C_vector[1]/E0;
	integrand+=2.0f*C_vector[2]*(E0-E_i)/E0_pow; E0_pow*=E0;
	integrand+=3.0f*C_vector[3]*(E1_pow+E1_prime_pow)/E0_pow; E0_pow*=E0; E1_pow*=E1; E1_prime_pow*=E1_prime;
	integrand+=4.0f*C_vector[4]*(E1_pow+E1_prime_pow)/E0_pow;
	integrand*=0.5f/E_i;
	return integrand;
}

__device__ float d_l_int_f(float EGamma,float T_r,float *D_vector)
{	float exp_EG=expf(EGamma/T_r)-1.0f;
	float integrand=(D_vector[0]+D_vector[1]/EGamma)/exp_EG;
	return integrand;
}

//Carry out an integral for the collisional excitation coefficient
__global__ void d_j_calc(double *d_params, double *E_j, double *B_vector, double *d_j_up, double *w, double *x)
{	
	//Calculate the integrand on each thread
	int node_num, integral_num;
	integral_num=threadIdx.x/d_datapoints;
	node_num=threadIdx.x % d_datapoints;
	integral_num+=blockIdx.x*d_block_mult;
	extern  __shared__ double d_j_up_temp[];	//Integrand is stored in shared memory
	//double *d_j_up_temp=&temp[0];
	double lim=(E_j[integral_num]+2.0*fabs(d_params[1])+ACC_J*d_params[0]);
	double E0=x[node_num]*lim+E_j[integral_num];
	double integrand=d_j_int(E0,E_j[integral_num],B_vector+integral_num*4)*w[node_num];
	double fermi=1.0/(1.0+exp((E0-d_params[1])/d_params[0]));
	double fermi_m=1.0/(1.0+exp((E0-E_j[integral_num]-d_params[1])/d_params[0]));
	d_j_up_temp[threadIdx.x]=integrand*fermi*(1.0-fermi_m);

	//Perform a standard reduction to integrate
    for (int d = d_datapoints>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (node_num<d)
	{
		d_j_up_temp[threadIdx.x]   += d_j_up_temp[threadIdx.x+d];
	}
    }
	__syncthreads();
    if (node_num==0) 
    {	
	d_j_up[integral_num]=d_j_up_temp[threadIdx.x]*lim;
    }
}

__global__ void d_j_calc_f(float *d_params, float *E_j, float *B_vector, float *d_j_up, float *w, float *x)
{	
	//Calculate the integrand on each thread
	int node_num, integral_num;
	integral_num=threadIdx.x/d_datapoints;
	node_num=threadIdx.x % d_datapoints;
	integral_num+=blockIdx.x*d_block_mult;
	extern  __shared__ float d_j_up_tempf[];	//Integrand is stored in shared memory
	//float *d_j_up_temp=&temp[0];
	float lim=(E_j[integral_num]+2.0f*fabsf(d_params[1])+ACC_Jf*d_params[0]);
	float E0=x[node_num]*lim+E_j[integral_num];
	float integrand=d_j_int_f(E0,E_j[integral_num],B_vector+integral_num*4)*w[node_num];
	float fermi=1.0f/(1.0f+expf((E0-d_params[1])/d_params[0]));
	float fermi_m=1.0f/(1.0f+expf((E0-E_j[integral_num]-d_params[1])/d_params[0]));
	d_j_up_tempf[threadIdx.x]=integrand*fermi*(1.0f-fermi_m);

	//Perform a standard reduction to integrate
    for (int d = d_datapoints>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (node_num<d)
	{
		d_j_up_tempf[threadIdx.x]   += d_j_up_tempf[threadIdx.x+d];
	}
    }
	__syncthreads();
    if (node_num==0) 
    {	
	d_j_up[integral_num]=d_j_up_tempf[threadIdx.x]*lim;
    }
}

//Carry out a double integral for the collisional ionization coefficient
//Involves now two reductions and the intermediate results stored in main GPU memory
__global__ void d_k_calc(double *d_params, double *E_i, double *C_vector,double *d_k_up, double *w, double *x)
{	
	//Calculate the integrand on each thread
	int node_num, integral_num;
	integral_num=threadIdx.x/d_datapoints;
	node_num=threadIdx.x % d_datapoints;
	integral_num+=blockIdx.x*d_block_mult;
	extern  __shared__ double d_k_up_temp[];
	//double *d_k_up_temp=&temp[0];
	double lim=(fabs(d_params[1])+50.0+ACC_K*d_params[0]);
	double E0prime=x[node_num]*lim;
	double E0=E0prime+E_i[integral_num];
	double fermiE0=1.0/(1.0+exp((E0-d_params[1])/d_params[0]));
	double E1, integrand, fermiE1, fermiE0prime, int_w_w=0;
	for (int idx=0;idx<d_datapoints;idx++)
	{
		E1=x[idx]*E0prime;
		integrand=d_k_int(E0,E1,E_i[integral_num],C_vector+integral_num*5);
		fermiE1=1.0/(1.0+exp((E1-d_params[1])/d_params[0]));
		fermiE0prime=1.0/(1.0+exp((E0prime-E1-d_params[1])/d_params[0]));
		int_w_w+=integrand*w[node_num]*w[idx]*E0prime*fermiE0*(1.0-fermiE1)*(1.0-fermiE0prime);
	}
	d_k_up_temp[threadIdx.x]=int_w_w;

 	//Perform a standard reduction to integrate
    for (int d = d_datapoints>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (node_num<d)
	{
		d_k_up_temp[threadIdx.x]   += d_k_up_temp[threadIdx.x+d];
	}
    }
	__syncthreads();
    if (node_num==0) 
    {	
	d_k_up[integral_num]=d_k_up_temp[threadIdx.x]*lim;
    }
}

__global__ void d_k_calc_f(float *d_params, float *E_i, float *C_vector,float *d_k_up, float *w, float *x)
{	
	//Calculate the integrand on each thread
	int node_num, integral_num;
	integral_num=threadIdx.x/d_datapoints;
	node_num=threadIdx.x % d_datapoints;
	integral_num+=blockIdx.x*d_block_mult;
	extern  __shared__ float d_k_up_tempf[];
	//float *d_k_up_temp=&temp[0];
	float lim=(fabsf(d_params[1])+50.0f+ACC_Kf*d_params[0]);
	float E0prime=x[node_num]*lim;
	float E0=E0prime+E_i[integral_num];
	float fermiE0=1.0f/(1.0f+exp((E0-d_params[1])/d_params[0]));
	float E1, integrand, fermiE1, fermiE0prime, int_w_w=0.0f;
	for (int idx=0;idx<d_datapoints;idx++)
	{
		E1=x[idx]*E0prime;
		integrand=d_k_int_f(E0,E1,E_i[integral_num],C_vector+integral_num*5);
		fermiE1=1.0f/(1.0f+exp((E1-d_params[1])/d_params[0]));
		fermiE0prime=1.0f/(1.0f+exp((E0prime-E1-d_params[1])/d_params[0]));
		int_w_w+=integrand*w[node_num]*w[idx]*E0prime*fermiE0*(1.0f-fermiE1)*(1.0f-fermiE0prime);
	}
	d_k_up_tempf[threadIdx.x]=int_w_w;

	//Series of reductions over the first integral
    for (int d = d_datapoints>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (node_num<d)
	{
		d_k_up_tempf[threadIdx.x]   += d_k_up_tempf[threadIdx.x+d];
	}
    }
	__syncthreads();
    if (node_num==0) 
    {	
	d_k_up[integral_num]=d_k_up_tempf[threadIdx.x]*lim;
    }
}

//Carry out an integral for the photoionization coefficient and energy change due to photoionization
//Single integral, similar to d_j_calc()
__global__ void d_l_calc(double *d_params, double *E_i, double *D_vector, double *d_l, double *d_le, double *w, double *x)
{	
	int node_num, integral_num;
	integral_num=threadIdx.x/d_datapoints;
	node_num=threadIdx.x % d_datapoints;
	integral_num+=blockIdx.x*d_block_mult;
	extern  __shared__ double d_l_temp[];
	//double *d_l_temp=&temp[0];
	double *d_le_temp=&d_l_temp[blockDim.x];
	double lim=ACC_L;
	if(d_params[1]>0.0){lim+=d_params[1];}
	double EGammaPrime=x[node_num]*lim;
	double EGamma=EGammaPrime+E_i[integral_num];
	double fermi_m=1.0-1.0/(1.0+exp((EGammaPrime-d_params[1])/d_params[0]));
	double integrand=d_l_int(EGamma,d_T_r,D_vector+integral_num*2)*w[node_num]*fermi_m;
	d_l_temp[threadIdx.x]=integrand;
	d_le_temp[threadIdx.x]=integrand*EGammaPrime;
    for (int d = d_datapoints>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (node_num<d)
	{
		d_l_temp[threadIdx.x]   += d_l_temp[threadIdx.x+d];
		d_le_temp[threadIdx.x]   += d_le_temp[threadIdx.x+d];
	}
    }
	__syncthreads();
    if (node_num==0) 
    {	
	d_l[integral_num]=d_l_temp[threadIdx.x]*lim;
	d_le[integral_num]=d_le_temp[threadIdx.x]*lim;
    }
}

__global__ void d_l_calc_f(float *d_params, float *E_i, float *D_vector, float *d_l, float *d_le, float *w, float *x)
{	
	int node_num, integral_num;
	integral_num=threadIdx.x/d_datapoints;
	node_num=threadIdx.x % d_datapoints;
	integral_num+=blockIdx.x*d_block_mult;
	extern  __shared__ float d_l_tempf[];
	//float *d_l_temp=&temp[0];
	float *d_le_temp=&d_l_tempf[blockDim.x];
	float lim=ACC_Lf;
	if(d_params[1]>0.0f){lim+=d_params[1];}
	float EGammaPrime=x[node_num]*lim;
	float EGamma=EGammaPrime+E_i[integral_num];
	float fermi_m=1.0f-1.0f/(1.0f+expf((EGammaPrime-d_params[1])/d_params[0]));
	float integrand=d_l_int_f(EGamma,d_T_r_f,D_vector+integral_num*2)*w[node_num]*fermi_m;
	d_l_tempf[threadIdx.x]=integrand;
	d_le_temp[threadIdx.x]=integrand*EGammaPrime;

    for (int d = d_datapoints>>1; d > 0; d >>= 1)
    {
      __syncthreads(); 
      if (node_num<d)
	{
		d_l_tempf[threadIdx.x]   += d_l_tempf[threadIdx.x+d];
		d_le_temp[threadIdx.x]   += d_le_temp[threadIdx.x+d];
	}
    }
	__syncthreads();
    if (node_num==0) 
    {	
	d_l[integral_num]=d_l_tempf[threadIdx.x]*lim;
	d_le[integral_num]=d_le_temp[threadIdx.x]*lim;
    }
}

void d_setup(double **d_params, double **d_B_vector, double **d_C_vector, double **d_D_vector, double **d_E_j, double **d_E_i, double **d_j, double **d_k, double **d_l, double **d_x, double **d_w, double *B_vector, double *C_vector, double *D_vector, double *E_j, double *E_i, double T_r, double *h_x, double *h_w, int ionizations_number, int excitations_number, int h_datapoints, hipStream_t *streams, int h_block_mult)
{
hipMalloc((void **)d_params,sizeof(double)*2);
hipMalloc((void **)d_B_vector,sizeof(double)*excitations_number*4);
hipMalloc((void **)d_C_vector,sizeof(double)*ionizations_number*5);
hipMalloc((void **)d_D_vector,sizeof(double)*ionizations_number*2);
hipMalloc((void **)d_E_j,sizeof(double)*excitations_number);
hipMalloc((void **)d_E_i,sizeof(double)*ionizations_number);
hipMalloc((void **)d_j,sizeof(double)*excitations_number);
hipMalloc((void **)d_k,sizeof(double)*ionizations_number);
hipMalloc((void **)d_l,2*sizeof(double)*ionizations_number);
hipMalloc((void **)d_x,sizeof(double)*h_datapoints);
hipMalloc((void **)d_w,sizeof(double)*h_datapoints);

hipMemcpyToSymbol(HIP_SYMBOL(d_ionizations_number),&ionizations_number,sizeof(ionizations_number));
hipMemcpyToSymbol(HIP_SYMBOL(d_excitations_number),&excitations_number,sizeof(excitations_number));
hipMemcpyToSymbol(HIP_SYMBOL(d_datapoints),&h_datapoints,sizeof(h_datapoints));
hipMemcpyToSymbol(HIP_SYMBOL(d_block_mult),&h_block_mult,sizeof(h_block_mult));
hipMemcpyToSymbol(HIP_SYMBOL(d_T_r),&T_r,sizeof(T_r));
hipMemcpy(*d_B_vector,B_vector,sizeof(double)*excitations_number*4,hipMemcpyHostToDevice);
hipMemcpy(*d_C_vector,C_vector,sizeof(double)*ionizations_number*5,hipMemcpyHostToDevice);
hipMemcpy(*d_D_vector,D_vector,sizeof(double)*ionizations_number*2,hipMemcpyHostToDevice);
hipMemcpy(*d_E_j,E_j,sizeof(double)*excitations_number,hipMemcpyHostToDevice);
hipMemcpy(*d_E_i,E_i,sizeof(double)*ionizations_number,hipMemcpyHostToDevice);
hipMemcpy(*d_x,h_x,sizeof(double)*h_datapoints,hipMemcpyHostToDevice);
hipMemcpy(*d_w,h_w,sizeof(double)*h_datapoints,hipMemcpyHostToDevice);

hipStreamCreate(&streams[0]);
hipStreamCreate(&streams[1]);
}

void d_setup_f(float **d_params, float **d_B_vector, float **d_C_vector, float **d_D_vector, float **d_E_j, float **d_E_i, float **d_j, float **d_k, float **d_l, float **d_x, float **d_w, float *B_vector, float *C_vector, float *D_vector, float *E_j, float *E_i, float T_r, float *h_x, float *h_w, int ionizations_number, int excitations_number, int h_datapoints, hipStream_t *streams, int h_block_mult)
{
hipMalloc((void **)d_params,sizeof(float)*2);
hipMalloc((void **)d_B_vector,sizeof(float)*excitations_number*4);
hipMalloc((void **)d_C_vector,sizeof(float)*ionizations_number*5);
hipMalloc((void **)d_D_vector,sizeof(float)*ionizations_number*2);
hipMalloc((void **)d_E_j,sizeof(float)*excitations_number);
hipMalloc((void **)d_E_i,sizeof(float)*ionizations_number);
hipMalloc((void **)d_j,sizeof(float)*excitations_number);
hipMalloc((void **)d_k,sizeof(float)*ionizations_number);
hipMalloc((void **)d_l,2*sizeof(float)*ionizations_number);
hipMalloc((void **)d_x,sizeof(float)*h_datapoints);
hipMalloc((void **)d_w,sizeof(float)*h_datapoints);

hipMemcpyToSymbol(HIP_SYMBOL(d_ionizations_number),&ionizations_number,sizeof(ionizations_number));
hipMemcpyToSymbol(HIP_SYMBOL(d_excitations_number),&excitations_number,sizeof(excitations_number));
hipMemcpyToSymbol(HIP_SYMBOL(d_datapoints),&h_datapoints,sizeof(h_datapoints));
hipMemcpyToSymbol(HIP_SYMBOL(d_block_mult),&h_block_mult,sizeof(h_block_mult));
hipMemcpyToSymbol(HIP_SYMBOL(d_T_r_f),&T_r,sizeof(T_r));
hipMemcpy(*d_B_vector,B_vector,sizeof(float)*excitations_number*4,hipMemcpyHostToDevice);
hipMemcpy(*d_C_vector,C_vector,sizeof(float)*ionizations_number*5,hipMemcpyHostToDevice);
hipMemcpy(*d_D_vector,D_vector,sizeof(float)*ionizations_number*2,hipMemcpyHostToDevice);
hipMemcpy(*d_E_j,E_j,sizeof(float)*excitations_number,hipMemcpyHostToDevice);
hipMemcpy(*d_E_i,E_i,sizeof(float)*ionizations_number,hipMemcpyHostToDevice);
hipMemcpy(*d_x,h_x,sizeof(float)*h_datapoints,hipMemcpyHostToDevice);
hipMemcpy(*d_w,h_w,sizeof(float)*h_datapoints,hipMemcpyHostToDevice);

hipStreamCreate(&streams[0]);
hipStreamCreate(&streams[1]);
}

void d_cleanup(double *d_params, double *d_B_vector, double *d_C_vector, double *d_E_j, double *d_E_i, double *d_j, double *d_k, double *d_l, double *d_x, double *d_w)
{
   hipFree(d_params);
   hipFree(d_B_vector);
   hipFree(d_C_vector);
   hipFree(d_E_j);
   hipFree(d_E_i);
   hipFree(d_j);
   hipFree(d_k);
   hipFree(d_l);
   hipFree(d_x);
   hipFree(d_w);
   hipDeviceReset();
}

void d_cleanup_f(float *d_params, float *d_B_vector, float *d_C_vector, float *d_E_j, float *d_E_i, float *d_j, float *d_k, float *d_l, float *d_x, float *d_w)
{
   hipFree(d_params);
   hipFree(d_B_vector);
   hipFree(d_C_vector);
   hipFree(d_E_j);
   hipFree(d_E_i);
   hipFree(d_j);
   hipFree(d_k);
   hipFree(d_l);
   hipFree(d_x);
   hipFree(d_w);
   hipDeviceReset();
}

void d_calculate_rates(double *d_params,double *d_B_vector, double *d_C_vector, double *d_D_vector, double *d_E_j, double *d_E_i, double *d_j, double *d_k, double *d_l, double *d_x, double *d_w,double *h_params, double *h_j,double *h_k,double *h_l,double *h_w, double *h_x, double  *ib_E, double n_e, double T_r, double *charge_vector, double *N, int states_number, int ionizations_number,int excitations_number,int h_datapoints,hipStream_t *streams, int h_block_mult)
{	
	hipMemcpy(d_params,h_params,sizeof(double)*2,hipMemcpyHostToDevice);
	d_j_calc<<<excitations_number/h_block_mult,h_datapoints*h_block_mult,h_datapoints*h_block_mult*sizeof(double),streams[0]>>>(d_params,d_E_j,d_B_vector,d_j,d_w,d_x);
	d_l_calc<<<ionizations_number/h_block_mult,h_datapoints*h_block_mult,2*h_datapoints*h_block_mult*sizeof(double),streams[0]>>>(d_params,d_E_i,d_D_vector,d_l,d_l+ionizations_number,d_w,d_x);
	d_k_calc<<<ionizations_number/h_block_mult,h_datapoints*h_block_mult,h_datapoints*h_block_mult*sizeof(double),streams[1]>>>(d_params,d_E_i,d_C_vector,d_k,d_w,d_x);
	*ib_E=h_ib_gauss_integration(h_params[0],n_e,h_params[1],T_r,states_number,charge_vector,N,h_datapoints,h_w,h_x);
	hipMemcpyAsync(h_j,d_j,sizeof(double)*excitations_number,hipMemcpyDeviceToHost,streams[0]);
	hipMemcpyAsync(h_l,d_l,2*sizeof(double)*ionizations_number,hipMemcpyDeviceToHost,streams[0]);
	hipMemcpyAsync(h_k,d_k,sizeof(double)*ionizations_number,hipMemcpyDeviceToHost,streams[1]);
	hipDeviceSynchronize();
}

void d_calculate_rates_f(float *d_params,float *d_B_vector, float *d_C_vector, float *d_D_vector, float *d_E_j, float *d_E_i, float *d_j, float *d_k, float *d_l, float *d_x, float *d_w,float *h_params, float *h_j,float *h_k,float *h_l,float *h_w, float *h_x, float  *ib_E, float n_e, float T_r, float *charge_vector, float *N, int states_number, int ionizations_number,int excitations_number,int h_datapoints,hipStream_t *streams, int h_block_mult)
{	
	hipMemcpy(d_params,h_params,sizeof(float)*2,hipMemcpyHostToDevice);
	d_j_calc_f<<<excitations_number/h_block_mult,h_datapoints*h_block_mult,h_datapoints*h_block_mult*sizeof(float),streams[0]>>>(d_params,d_E_j,d_B_vector,d_j,d_w,d_x);
	d_l_calc_f<<<ionizations_number/h_block_mult,h_datapoints*h_block_mult,2*h_datapoints*h_block_mult*sizeof(float),streams[0]>>>(d_params,d_E_i,d_D_vector,d_l,d_l+ionizations_number,d_w,d_x);
	d_k_calc_f<<<ionizations_number/h_block_mult,h_datapoints*h_block_mult,h_datapoints*h_block_mult*sizeof(float),streams[1]>>>(d_params,d_E_i,d_C_vector,d_k,d_w,d_x);
	*ib_E=h_ib_gauss_integration_f(h_params[0],n_e,h_params[1],T_r,states_number,charge_vector,N,h_datapoints,h_w,h_x);
	hipMemcpyAsync(h_j,d_j,sizeof(float)*excitations_number,hipMemcpyDeviceToHost,streams[0]);
	hipMemcpyAsync(h_l,d_l,2*sizeof(float)*ionizations_number,hipMemcpyDeviceToHost,streams[0]);
	hipMemcpyAsync(h_k,d_k,sizeof(float)*ionizations_number,hipMemcpyDeviceToHost,streams[1]);
	hipDeviceSynchronize();
}

//This is the RK-4 solver using GPUs to calculate atomic rates
//See h_solve_RK4 in Rate_Functions_CPU.c for more details
void hd_solve_RK4(int states_number, int ionizations_number, int excitations_number, double delta, double *charge_vector, double *N, double *N_temp1, double *N_temp2, double *N_temp3, double *N_temp4, double *IntE_temp, double *n_e, double *T_e, double *T_F, double *Internal_Energy,int h_datapoints, double *h_w, double *h_x, double *h_j, double *h_k, double *h_l, double T_r, int *excitations_indices, int *ionizations_indices, double *E_i, double *E_j,double *A_vector, double *B_vector, double *C_vector, double *D_vector, double *R_1, double *R_2, double *h_params, double *d_params, double *d_B_vector, double *d_C_vector, double *d_D_vector, double *d_E_j, double *d_E_i, double *d_j, double *d_k, double *d_l, double *d_x, double *d_w,hipStream_t *streams, int h_block_mult)
{	//RK-4 method, N_t[n+1] = N_t[n]+Delta_t/6*(k_1+2k_2+2k_3+k_4)
	//Declare variables, copy current values to temporary arrays
	double n_e_temp, T_F_temp, T_e_temp, delta_2=0.5*delta, Int_energy_temp1, Int_energy_temp2, Int_energy_temp3, Int_energy_temp4, ib_E;
	cblas_dcopy(states_number,N,1,N_temp1,1);
	cblas_dcopy(states_number,N,1,N_temp2,1);
	cblas_dcopy(states_number,N,1,N_temp3,1);
	cblas_dcopy(states_number,N,1,N_temp4,1);

	//Coefficients 1
	h_params[0]=*T_e;
	h_params[1]=Get_Chemical_Potential(*T_e,*T_F);
	d_calculate_rates(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,*n_e,T_r,charge_vector,N, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices(states_number,ionizations_number,excitations_number,*T_e,*n_e,h_params[1] ,R_1,R_2,h_j,h_k, h_l,excitations_indices, ionizations_indices,E_i,E_j);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_1,states_number,N,1,1.0,N_temp1,1);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_2,states_number,N,1,0.0,IntE_temp,1);
	Int_energy_temp1=vector_sum(states_number,IntE_temp)+*Internal_Energy+ib_E*delta_2;

	//Coefficients 2
	n_e_temp=Get_n_e(states_number,N_temp1,charge_vector);
	T_F_temp=Fermi_Energy(n_e_temp);
	T_e_temp=T_F_temp*Invert_C_V(Int_energy_temp1/(T_F_temp*n_e_temp));
	h_params[0]=T_e_temp;
	h_params[1]=Get_Chemical_Potential(T_e_temp,T_F_temp);
	d_calculate_rates(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,n_e_temp,T_r,charge_vector,N_temp1, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices(states_number,ionizations_number,excitations_number,T_e_temp,n_e_temp,h_params[1],R_1,R_2,h_j,h_k,h_l,excitations_indices,ionizations_indices,E_i,E_j);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_1,states_number,N_temp1,1,1.0,N_temp2,1);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_2,states_number,N_temp1,1,0.0,IntE_temp,1);
	Int_energy_temp2=vector_sum(states_number,IntE_temp)+*Internal_Energy+ib_E*delta_2;

	//Coefficients 3
	n_e_temp=Get_n_e(states_number,N_temp2,charge_vector);
	T_F_temp=Fermi_Energy(n_e_temp);
	T_e_temp=T_F_temp*Invert_C_V(Int_energy_temp2/(T_F_temp*n_e_temp));
	h_params[0]=T_e_temp;
	h_params[1]=Get_Chemical_Potential(T_e_temp,T_F_temp);
	d_calculate_rates(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,n_e_temp,T_r,charge_vector,N_temp2, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices(states_number,ionizations_number,excitations_number,T_e_temp,n_e_temp,h_params[1],R_1,R_2,h_j,h_k,h_l,excitations_indices,ionizations_indices,E_i,E_j);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta,R_1,states_number,N_temp2,1,1.0,N_temp3,1);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta,R_2,states_number,N_temp2,1,0.0,IntE_temp,1);
	Int_energy_temp3=vector_sum(states_number,IntE_temp)+*Internal_Energy+ib_E*delta;

	//Coefficients 4
	n_e_temp=Get_n_e(states_number,N_temp3,charge_vector);
	T_F_temp=Fermi_Energy(n_e_temp);
	T_e_temp=T_F_temp*Invert_C_V(Int_energy_temp3/(T_F_temp*n_e_temp));
	h_params[0]=T_e_temp;
	h_params[1]=Get_Chemical_Potential(T_e_temp,T_F_temp);
	d_calculate_rates(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,n_e_temp,T_r,charge_vector,N_temp3, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices(states_number,ionizations_number,excitations_number,T_e_temp,n_e_temp,h_params[1],R_1,R_2,h_j,h_k,h_l,excitations_indices,ionizations_indices,E_i,E_j);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_1,states_number,N_temp3,1,-1.0,N_temp4,1);
	cblas_dgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_2,states_number,N_temp2,1,0.0,IntE_temp,1);
	Int_energy_temp4=vector_sum(states_number,IntE_temp)-*Internal_Energy+ib_E*delta_2;

	//Calculate starting values for next iteration
	cblas_daxpy(states_number,1.0,N_temp1,1,N_temp3,1);
	cblas_daxpy(states_number,2.0,N_temp2,1,N_temp4,1);
	cblas_daxpy(states_number,1.0,N_temp3,1,N_temp4,1);
 	cblas_dscal(states_number,0.3333333333333333,N_temp4,1);
	cblas_dcopy(states_number,N_temp4,1,N,1);
	*Internal_Energy=(Int_energy_temp1+2.0*Int_energy_temp2+Int_energy_temp3+Int_energy_temp4)*0.3333333333333333;
	*n_e=Get_n_e(states_number,N,charge_vector);
	*T_F=Fermi_Energy(*n_e);
	*T_e=*T_F*Invert_C_V(*Internal_Energy/(*T_F* *n_e));
}

void hd_solve_RK4_f(int states_number, int ionizations_number, int excitations_number, float delta, float *charge_vector, float *N, float *N_temp1, float *N_temp2, float *N_temp3, float *N_temp4, float *IntE_temp, float *n_e, float *T_e, float *T_F, float *Internal_Energy,int h_datapoints, float *h_w, float *h_x, float *h_j, float *h_k, float *h_l, float T_r, int *excitations_indices, int *ionizations_indices, float *E_i, float *E_j,float *A_vector, float *B_vector, float *C_vector, float *D_vector, float *R_1, float *R_2, float *h_params, float *d_params, float *d_B_vector, float *d_C_vector, float *d_D_vector, float *d_E_j, float *d_E_i, float *d_j, float *d_k, float *d_l, float *d_x, float *d_w,hipStream_t *streams, int h_block_mult)
{	//RK-4 method, N_t[n+1] = N_t[n]+Delta_t/6*(k_1+2k_2+2k_3+k_4)
	//Declare variables, copy current values to temporary arrays
	float n_e_temp, T_F_temp, T_e_temp, delta_2=0.5f*delta, Int_energy_temp1, Int_energy_temp2, Int_energy_temp3, Int_energy_temp4, ib_E;
	cblas_scopy(states_number,N,1,N_temp1,1);
	cblas_scopy(states_number,N,1,N_temp2,1);
	cblas_scopy(states_number,N,1,N_temp3,1);
	cblas_scopy(states_number,N,1,N_temp4,1);

	//Coefficients 1
	h_params[0]=*T_e;
	h_params[1]=Get_Chemical_Potential(*T_e,*T_F);
	d_calculate_rates_f(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,*n_e,T_r,charge_vector,N, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices_f(states_number,ionizations_number,excitations_number,*T_e,*n_e,h_params[1] ,R_1,R_2,h_j,h_k, h_l,excitations_indices, ionizations_indices,E_i,E_j);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_1,states_number,N,1,1.0f,N_temp1,1);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_2,states_number,N,1,0.0f,IntE_temp,1);
	Int_energy_temp1=vector_sum_f(states_number,IntE_temp)+*Internal_Energy+ib_E*delta_2;

	//Coefficients 2
	n_e_temp=Get_n_e_f(states_number,N_temp1,charge_vector);
	T_F_temp=Fermi_Energy_f(n_e_temp);
	T_e_temp=T_F_temp*Invert_C_V_f(Int_energy_temp1/(T_F_temp*n_e_temp));
	h_params[0]=T_e_temp;
	h_params[1]=Get_Chemical_Potential_f(T_e_temp,T_F_temp);
	d_calculate_rates_f(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,n_e_temp,T_r,charge_vector,N_temp1, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices_f(states_number,ionizations_number,excitations_number,T_e_temp,n_e_temp,h_params[1],R_1,R_2,h_j,h_k,h_l,excitations_indices,ionizations_indices,E_i,E_j);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_1,states_number,N_temp1,1,1.0f,N_temp2,1);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_2,states_number,N_temp1,1,0.0f,IntE_temp,1);
	Int_energy_temp2=vector_sum_f(states_number,IntE_temp)+*Internal_Energy+ib_E*delta_2;

	//Coefficients 3
	n_e_temp=Get_n_e_f(states_number,N_temp2,charge_vector);
	T_F_temp=Fermi_Energy_f(n_e_temp);
	T_e_temp=T_F_temp*Invert_C_V_f(Int_energy_temp2/(T_F_temp*n_e_temp));
	h_params[0]=T_e_temp;
	h_params[1]=Get_Chemical_Potential_f(T_e_temp,T_F_temp);
	d_calculate_rates_f(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,n_e_temp,T_r,charge_vector,N_temp2, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices_f(states_number,ionizations_number,excitations_number,T_e_temp,n_e_temp,h_params[1],R_1,R_2,h_j,h_k,h_l,excitations_indices,ionizations_indices,E_i,E_j);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta,R_1,states_number,N_temp2,1,1.0f,N_temp3,1);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta,R_2,states_number,N_temp2,1,0.0f,IntE_temp,1);
	Int_energy_temp3=vector_sum_f(states_number,IntE_temp)+*Internal_Energy+ib_E*delta;

	//Coefficients 4
	n_e_temp=Get_n_e_f(states_number,N_temp3,charge_vector);
	T_F_temp=Fermi_Energy_f(n_e_temp);
	T_e_temp=T_F_temp*Invert_C_V_f(Int_energy_temp3/(T_F_temp*n_e_temp));
	h_params[0]=T_e_temp;
	h_params[1]=Get_Chemical_Potential_f(T_e_temp,T_F_temp);
	d_calculate_rates_f(d_params,d_B_vector, d_C_vector,d_D_vector, d_E_j, d_E_i, d_j, d_k, d_l, d_x,  d_w,h_params,h_j,h_k,h_l,h_w,h_x,&ib_E,n_e_temp,T_r,charge_vector,N_temp3, states_number, ionizations_number,excitations_number,h_datapoints,streams,h_block_mult);
	h_create_rate_matrices_f(states_number,ionizations_number,excitations_number,T_e_temp,n_e_temp,h_params[1],R_1,R_2,h_j,h_k,h_l,excitations_indices,ionizations_indices,E_i,E_j);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_1,states_number,N_temp3,1,-1.0f,N_temp4,1);
	cblas_sgemv(CblasRowMajor,CblasNoTrans,states_number,states_number,delta_2,R_2,states_number,N_temp2,1,0.0f,IntE_temp,1);
	Int_energy_temp4=vector_sum_f(states_number,IntE_temp)-*Internal_Energy+ib_E*delta_2;

	//Calculate starting values for next iteration
	cblas_saxpy(states_number,1.0f,N_temp1,1,N_temp3,1);
	cblas_saxpy(states_number,2.0f,N_temp2,1,N_temp4,1);
	cblas_saxpy(states_number,1.0f,N_temp3,1,N_temp4,1);
 	cblas_sscal(states_number,0.3333333333333333f,N_temp4,1);
	cblas_scopy(states_number,N_temp4,1,N,1);
	*Internal_Energy=(Int_energy_temp1+2.0f*Int_energy_temp2+Int_energy_temp3+Int_energy_temp4)*0.3333333333333333f;
	*n_e=Get_n_e_f(states_number,N,charge_vector);
	*T_F=Fermi_Energy_f(*n_e);
	*T_e=*T_F*Invert_C_V_f(*Internal_Energy/(*T_F* *n_e));
}
